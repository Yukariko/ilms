#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.	All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

// System includes
#include <stdio.h>
#include <assert.h>
#include <iostream>

// CUDA runtime
#include <hip/hip_runtime.h>

#include "bloomfilter.h"

//const int bf_size = 2 * 1024 * 1024;

__global__ void cudaSetBitArray(unsigned char *filter, long long *bitArray)
{
	int tid = blockIdx.x;
	filter[bitArray[tid] >> 3] |= (1 << (bitArray[tid] & 7));
}

__global__ void cudaLookBitArray(unsigned char *filter, long long *bitArray, int *res)
{
	int tid = blockIdx.x;
	if(!(filter[bitArray[tid] >> 3] & (1 << (bitArray[tid] & 7))))
		*res = 0;
}

__global__ void cudaLookFilters(unsigned char **filters, long long *bitArray, unsigned char *ans)
{
	int nFilter = blockIdx.x * threadIdx.x;
	int nHash = threadIdx.y;

	if(!(filters[nFilter][bitArray[nHash] >> 3] & (1 << (bitArray[nHash] & 7))))
		ans[nFilter] = 0;
}

Bloomfilter::Bloomfilter(long long size, int numHash,long long (**hash)(char *))
{
	this->size = size;
	this->numHash = numHash;

	this->hash = (long long (**)(char *))malloc(sizeof(long long (*)(char *)) * numHash);

	for(int i=0;i<numHash;i++)
		this->hash[i] = hash[i];

	error_handling( hipMalloc((void **)&filter, size / 8 + 1) );
	error_handling( hipMemset((void *)filter, 0, size / 8 + 1) );
	error_handling( hipMalloc((void **)&cudaBitArray, sizeof(long long) * numHash) );
	error_handling( hipMalloc((void **)&cudaRes, sizeof(int)) );
}

Bloomfilter::~Bloomfilter()
{
		hipFree(cudaBitArray);
		hipFree(filter);
		hipFree(cudaRes);
}

void Bloomfilter::insert(char *data)
{
	long long *bitArray;
	getBitArray(bitArray, data);
	cudaSetBitArray<<<numHash,1>>>(filter, cudaBitArray);
}

bool Bloomfilter::lookup(char *data)
{
	long long *bitArray;
	getBitArray(bitArray, data);
	return lookBitArray(bitArray);
}

unsigned char *Bloomfilter::getFilter()
{
	return filter;
}

void Bloomfilter::copyFilter(unsigned char *hostFilter)
{
	return error_handling( hipMemcpy((void *)hostFilter, (const void *)filter, size / 8 + 1, hipMemcpyDeviceToHost) );
}

void Bloomfilter::getBitArray(long long *&bitArray, char *data)
{
	long long array[20];
	for(int i=0;i<numHash;i++)
	{
		array[i] = hash[i](data) % size;
	}
	error_handling( hipMemcpy((void *)cudaBitArray, (const void *)array, sizeof(long long) * numHash, hipMemcpyHostToDevice) );
	//error_handling( hipMemcpy((void *)bitArray, (const void *)cudaBitArray, sizeof(long long) * numHash, hipMemcpyHostToDevice) );
	bitArray = cudaBitArray;
}

bool Bloomfilter::lookBitArray(long long *bitArray)
{
	int res = 1;
	error_handling( hipMemcpy((void *)cudaRes, (const void *)&res, sizeof(int), hipMemcpyHostToDevice) );

	cudaLookBitArray<<<numHash,1>>>(filter, bitArray, cudaRes);

	error_handling( hipMemcpy((void *)&res, (const void *)cudaRes,sizeof(int), hipMemcpyDeviceToHost) );
	return !!res;
}

void Bloomfilter::initFilters(unsigned char ***filters, unsigned int size)
{
	error_handling( hipMalloc((void **)filters, size * sizeof(unsigned char *)) );
}

void Bloomfilter::insertFilters(unsigned char **filters, unsigned int idx)
{
	error_handling( hipMemcpy(&filters[idx], (const void *)&filter, sizeof(unsigned char *), hipMemcpyHostToDevice) );
}

void Bloomfilter::initAnswer(unsigned char **ans, unsigned int size)
{
	error_handling( hipMalloc((void **)ans, size) );
}

void Bloomfilter::setAnswer(unsigned char *ans, unsigned int size)
{
	error_handling( hipMemset((void *)ans, 1, size) );
}

void Bloomfilter::lookFilters(unsigned char **filters, unsigned char *cuda_ans, long long *bitArray, unsigned char *ans, unsigned int size)
{
	setAnswer(cuda_ans, size);

	dim3 block(32);
	dim3 thread(size / 32, 11);

	cudaLookFilters<<<block, thread>>>(filters, bitArray, cuda_ans);
	error_handling( hipMemcpy((void *)ans, (const void *)cuda_ans, size, hipMemcpyDeviceToHost) );
}


void Bloomfilter::error_handling(hipError_t n)
{
	if(n)
	{
		printf("Error! %d\n",n);
		exit(0);
	}
}
