#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.	All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

// System includes
#include <stdio.h>
#include <assert.h>
#include <iostream>

// CUDA runtime
#include <hip/hip_runtime.h>

#include "bloomfilter.h"

using namespace std;

//const int bf_size = 2 * 1024 * 1024;

__global__ void cudaSetBitArray(unsigned char *filter, long long *bitArray)
{
	int tid = blockIdx.x;
	filter[bitArray[tid] >> 3] |= (1 << (bitArray[tid] & 7));
}

__global__ void cudaLookBitArray(unsigned char *filter, long long *bitArray, int *res)
{
	int tid = blockIdx.x;
	if(!(filter[bitArray[tid] >> 3] & (1 << (bitArray[tid] & 7))))
		*res = 0;
}

__global__ void cudaLookFilters(unsigned char **filters, long long *bitArray, unsigned char *ans)
{
	int nFilter = threadIdx.x;
	int nHash = threadIdx.y;
	
	if(!(filters[nFilter][bitArray[nHash] >> 3] & (1 << (bitArray[nHash] & 7))))
		ans[nFilter] = 0;
}

__global__ void cudaMergeFilter(unsigned char *dstFilter, unsigned char *srcFilter)
{
	int idx = blockIdx.x * 1024 + threadIdx.x;
	dstFilter[idx] |= srcFilter[idx];
}

Bloomfilter::Bloomfilter(long long size, int numHash,long long (**hash)(const char *))
{
	this->size = size;
	this->numHash = numHash;

	this->hash = (long long (**)(const char *))malloc(sizeof(long long (*)(const char *)) * numHash);

	for(int i=0;i<numHash;i++)
		this->hash[i] = hash[i];

	error_handling( hipMalloc((void **)&filter, size / 8 + 1) );
	error_handling( hipMemset((void *)filter, 0, size / 8 + 1) );
	error_handling( hipMalloc((void **)&cudaBitArray, sizeof(long long) * numHash) );
	error_handling( hipMalloc((void **)&cudaRes, sizeof(int)) );
}

Bloomfilter::~Bloomfilter()
{
		hipFree(cudaBitArray);
		hipFree(filter);
		hipFree(cudaRes);
}

void Bloomfilter::insert(const char *data)
{
	long long *bitArray;
	getBitArray(bitArray, data);
	cudaSetBitArray<<<numHash,1>>>(filter, cudaBitArray);
}

bool Bloomfilter::lookup(const char *data)
{
	long long *bitArray;
	getBitArray(bitArray, data);
	return lookBitArray(bitArray);
}

unsigned char *Bloomfilter::getFilter()
{
	return filter;
}

void Bloomfilter::copyFilter(unsigned char *hostFilter)
{
	return error_handling( hipMemcpy((void *)hostFilter, (const void *)filter, size / 8 + 1, hipMemcpyDeviceToHost) );
}

void Bloomfilter::setFilter(unsigned char *hostFilter)
{
	return error_handling( hipMemcpy((void *)filter, (const void *)hostFilter, size / 8 + 1, hipMemcpyHostToDevice) );
}

void Bloomfilter::getBitArray(long long *&bitArray, const char *data)
{
	long long array[20];
	for(int i=0;i<numHash;i++)
	{
		array[i] = hash[i](data) % size;
	}
	error_handling( hipMemcpy((void *)cudaBitArray, (const void *)array, sizeof(long long) * numHash, hipMemcpyHostToDevice) );
	bitArray = cudaBitArray;
}

bool Bloomfilter::lookBitArray(long long *bitArray)
{
	int res = 1;
	error_handling( hipMemcpy((void *)cudaRes, (const void *)&res, sizeof(int), hipMemcpyHostToDevice) );

	cudaLookBitArray<<<numHash,1>>>(filter, bitArray, cudaRes);

	error_handling( hipMemcpy((void *)&res, (const void *)cudaRes,sizeof(int), hipMemcpyDeviceToHost) );
	return !!res;
}

void Bloomfilter::initFilters(unsigned char ***filters, unsigned int size)
{
	error_handling( hipMalloc((void **)filters, size * sizeof(unsigned char *)) );
}

void Bloomfilter::insertFilters(unsigned char **filters, unsigned int idx)
{
	//cout << filters << endl;
	//cout << filters + idx << endl;
	error_handling( hipMemcpy((void **)(filters + idx), (const void *)&filter, sizeof(unsigned char *), hipMemcpyHostToDevice) );
}

void Bloomfilter::initAnswer(unsigned char **ans, unsigned int size)
{
	error_handling( hipMalloc((void **)ans, size) );
}

void Bloomfilter::setAnswer(unsigned char *ans, unsigned int size)
{
	error_handling( hipMemset((void *)ans, 1, size) );
}

void Bloomfilter::lookFilters(unsigned char **filters, unsigned char *cuda_ans, long long *bitArray, unsigned char *ans, unsigned int size)
{
	setAnswer(cuda_ans, size);

	dim3 block(1);
	dim3 thread(size,11);

	cudaLookFilters<<<block, thread>>>(filters, bitArray, cuda_ans);
	error_handling( hipMemcpy((void *)ans, (const void *)cuda_ans, size, hipMemcpyDeviceToHost) );
}

void Bloomfilter::zeroFilter()
{
	error_handling( hipMemset((void *)filter, 0, size / 8 + 1) );
}

void Bloomfilter::mergeFilter(unsigned char *filter)
{
	dim3 block(1024*2);
	dim3 thread(1024);
	cudaMergeFilter<<<block, thread>>>(this->filter, filter);
}


void Bloomfilter::error_handling(hipError_t n)
{
	if(n)
	{
		printf("Error! %d\n",n);
		exit(0);
	}
}
